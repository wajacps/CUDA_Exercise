
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>

// Add your kernel here
__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

// main
int main(void)
{
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	// Allocate memory in Device
	hipMalloc ((void **) &d_a, size);
	hipMalloc ((void **) &d_b, size);
	hipMalloc ((void **) &d_c, size);

	// Initialize value
	a = 2;
	b = 7;

	// Copy data from Host to Device
	hipMemcpy (d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy (d_b, &b, size, hipMemcpyHostToDevice);

	// Execute
	add<<<1,1>>>(d_a, d_b, d_c);

	// Copy result back to Host
	// Take note that it will be smart enough to wait
	// until the task at device completed
	hipMemcpy (&c, d_c, size, hipMemcpyDeviceToHost);

	// Clean up
	hipFree (d_a);
	hipFree (d_b);
	hipFree (d_c);

	printf("Task Completed: c = %d + %d = %d\n" ,a, b, c);

	return 0;
}
